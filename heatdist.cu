/*
 *  Please write your name and net ID below
 *  
 *  Last name: Boran
 *  First name: Tudor
 *  Net ID: N13059231
 * 
 *  I have attached a readme, you can also compile with make (which I used to get this in nsight, because I love IDEs)
 */


/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s) that you need to write too. 
 * 
 * You compile with:
 * 		nvcc -o heatdist -arch=sm_60 heatdist.cu   
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

/* Tile size */
#define TILE_SIZE 8

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void  seq_heat_dist(float *, unsigned int, unsigned int);
void  gpu_heat_dist(float *, unsigned int, unsigned int);
__global__ void  gpu_kernel(float *, float *, unsigned int);
void  check_err(hipError_t, char *);

/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char * argv[])
{
  unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = 0; // CPU or GPU
  int iterations = 0;
  int i;
  
  /* The 2D array of points will be treated as 1D array of NxN elements */
  float * playground; 
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }
  
  type_of_device = atoi(argv[3]);
  N = (unsigned int) atoi(argv[1]);
  iterations = (unsigned int) atoi(argv[2]);
 
  
  /* Dynamically allocate NxN array of floats */
  playground = (float *)calloc(N*N, sizeof(float));
  if( !playground )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
   exit(1);
  }
  
  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements to 70F
  for(i = 0; i < N; i++)
    playground[index(0,i,N)] = 70;
    
  for(i = 0; i < N; i++)
    playground[index(i,0,N)] = 70;
  
  for(i = 0; i < N; i++)
    playground[index(i,N-1, N)] = 70;
  
  for(i = 0; i < N; i++)
    playground[index(N-1,i,N)] = 70;
  
  // from (0,10) to (0,30) inclusive are 100F
  for(i = 10; i <= 30; i++)
    playground[index(0,i,N)] = 100;
  
   // from (n-1,10) to (n-1,30) inclusive are 150F
  for(i = 10; i <= 30; i++)
    playground[index(N-1,i,N)] = 150;
  
  if( !type_of_device ) // The CPU sequential version
  {  
    start = clock();
    seq_heat_dist(playground, N, iterations);
    end = clock();
  }
  else  // The GPU version
  {
     start = clock();
     gpu_heat_dist(playground, N, iterations); 
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);
  
  free(playground);
  
  return 0;

}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
  // Loop indices
  int i, j, k;
  int upper = N-1;
  
  // number of bytes to be copied between array temp and array playground
  unsigned int num_bytes = 0;
  
  float * temp; 
  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  temp = (float *)calloc(N*N, sizeof(float));
  if( !temp )
  {
   fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
   exit(1);
  }
  
  num_bytes = N*N*sizeof(float);
  
  /* Copy initial array in temp */
  memcpy((void *)temp, (void *) playground, num_bytes);
  
  for( k = 0; k < iterations; k++)
  {
    /* Calculate new values and store them in temp */
    for(i = 1; i < upper; i++)
      for(j = 1; j < upper; j++)
	temp[index(i,j,N)] = (playground[index(i-1,j,N)] + 
	                      playground[index(i+1,j,N)] + 
			      playground[index(i,j-1,N)] + 
			      playground[index(i,j+1,N)])/4.0;
  
			      
   			      
    /* Move new values into old values */ 
    memcpy((void *)playground, (void *) temp, num_bytes);
  }
  
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{


	// number of bytes to be copied between array temp and array playground
	size_t count = N*N;
	unsigned int num_bytes = count*sizeof(float);
	unsigned int i;

	float *d_temp = NULL, *d_playground = NULL;

	/* Dynamically allocate another array for temp values */
	/* Dynamically allocate NxN array of floats */
	hipError_t err;
	err = hipMalloc((void**)&d_temp, num_bytes);
	err = hipMalloc((void**)&d_playground, num_bytes);
	check_err(err, "allocating memory on device.");

	err = hipMemcpy(d_temp, playground, count*sizeof(float), hipMemcpyHostToDevice);
	err = hipMemcpy(d_playground, playground, count*sizeof(float), hipMemcpyHostToDevice);
	check_err(err, "copying array to device memory.");

	dim3 block(TILE_SIZE, TILE_SIZE, 1);
	dim3 grid(N/TILE_SIZE, N/TILE_SIZE, 1);
	for (i = 0; i < iterations; i++){
		gpu_kernel<<<grid, block>>>(d_playground, d_temp, N);
		err = hipMemcpy(d_playground, d_temp, count*sizeof(float), hipMemcpyDeviceToDevice);
		check_err(err, "syncing array");
	}

	err = hipMemcpy(playground, d_playground, count*sizeof(float), hipMemcpyDeviceToHost);
	check_err(err, "copying array back to host.");
}

__global__
void gpu_kernel(float *d_playground, float *d_temp, unsigned int N)
{
	unsigned int upper = N;
	unsigned int i, j;
	i = blockIdx.x*blockDim.x + threadIdx.x;
	j = blockIdx.y*blockDim.y + threadIdx.y;

	if (i > 0 && i < upper && j > 0 && j < upper)
	{
		d_temp[index(i,j,N)] = (d_playground[index(i-1,j,N)] +
				d_playground[index(i+1,j,N)] +
				d_playground[index(i,j-1,N)] +
				d_playground[index(i,j+1,N)])/4.0;
	}
}

void  check_err(hipError_t err, char *msg)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "CUDA Error: %s\n", msg);
		exit(1);
	}
}


